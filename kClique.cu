#include "hip/hip_runtime.h"
#include<bits/stdc++.h>
#include<thrust/device_vector.h>
using namespace std;

bool *adjacencyMatrix;
int *degree;

int Count[1];

__device__ int glock=0;

__global__ void kclique(int* ddegree,int start, int presentNodes, int K, int N,bool* adjacencyMat,int *KCount)
{
    int tid=threadIdx.x;
    int bid=blockIdx.x;
    
    int t = bid*128+tid;
    
    if(t<N)
    {
        int st[10000][4];
        int top=-1;
        int dclique[10000];
       
        top++;
        
        dclique[1]=t+1;
        st[top][0]=t+2,st[top][1]=1,st[top][2]=2,st[top][3]=K;
        
        KCount[0]=0;
        while(top!=-1)
        {
            int j=st[top][0],i=st[top][1],l=st[top][2],s=st[top][3];


            top--;
            if(j+1<=N)
            {
                top++;
                st[top][0]=j+1,st[top][1]=i,st[top][2]=l,st[top][3]=s;
            }
            if(ddegree[j]>=s-1)
            {

                dclique[l]=j;
                bool flag=true;
                for(int x=1;x<l+1;x++)
                {
                    for(int y=x+1;y<l+1;y++)
                    {
                        if(adjacencyMat[ 1ll*dclique[x]*1000000+dclique[y] ]==false)
                        {    
                            flag=false;
                            break;
                        }
                    }
                    if(!flag)
                        break;
                }
                if(flag)
                {
                    if(l<s)
                    {

                        top++;
                        st[top][0]=j+1,st[top][1]=j+1,st[top][2]=l+1,st[top][3]=s;
                    }
                    else
                    {
                        while(atomicCAS(&glock,0,1)) {}
                        __threadfence();   

                        KCount[0]++;

                        __threadfence();
                        atomicExch(&glock,0);

                    }
                }
            }
        }
    }
    __syncthreads();
}

int main()
{
    int k;
    string path;
   
    cin>>path;
    cin>>k;

    degree = (int*)malloc(sizeof(int)*1000000);
    hipMallocManaged(&adjacencyMatrix, 1000000000000*sizeof(bool));
    
    ifstream MyReadFile(path);
    string myText;
    int n=0;
    while (getline (MyReadFile, myText)){
        int a,b,i=0;
        string t="";
        while(myText[i]!=' ')
        {
            t+=myText[i];
            i++;
        }
        a=stoi(t);
        b=stoi(myText.substr(i+1));

        n = max(n,max(a,b));

        adjacencyMatrix[1ll*a*1000000+b] = true;
        adjacencyMatrix[1ll*b*1000000+a] = true;
        degree[a]++;
        degree[b]++;
    }

    int *ddegree,*KCount;
    hipMalloc((void**)&ddegree, 10000*sizeof(int));
    hipMalloc((void**)&KCount, 1*sizeof(int));
    
    hipMemcpy(ddegree, degree, 10000*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(KCount, Count, 1*sizeof(int), hipMemcpyHostToDevice);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float gpu_time = 0.0f;
    
    hipEventRecord(start, 0); 
    kclique<<<41,128>>>(ddegree,0,1,k,n,adjacencyMatrix,KCount);
    hipEventRecord(stop, 0); 
    hipEventSynchronize(stop);
    
    hipEventElapsedTime(&gpu_time, start, stop);
    
    hipMemcpy(Count, KCount, 1*sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    cout<<Count[0]<<endl;
    cout<<"Execution Time: "<<gpu_time<<" ms"<<endl;
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    hipFree(ddegree);
    hipFree(KCount);

    free(degree);

    return 0;
}




